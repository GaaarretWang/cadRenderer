#include "hip/hip_runtime.h"
#include "fixDepth.h"

__global__ static void convert_4_to_channels(int w, int h, unsigned short * depth_pixels_cuda) {  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    int idy = blockIdx.y * blockDim.y + threadIdx.y;  

    if (idx < w && idy < h) {  
        unsigned short cur_depth_color = depth_pixels_cuda[w * idy + idx];
        if(cur_depth_color > 100)
            return;
        else{
            unsigned short count = 0;
            unsigned short total = 0;
            for(int m = -1; m <= 1; m ++){ //height
                for(int n = -1; n <= 1; n ++){ //width
                    if((idy + m) >= 0 && (idy + m) < h && (idx + n) >= 0 && (idx + n) < w){
                        unsigned short neighbor_pixel = depth_pixels_cuda[w * (idy + m) + idx + n];
                        
                        if(neighbor_pixel > 100){
                            total += neighbor_pixel;
                            count ++;
                        }
                    }
                }
            }
            if(count > 0)
                depth_pixels_cuda[w * idy + idx] = total / count;
            return;
        }

    }  
}  

unsigned short * depth_pixels_cuda;

void allocate_fix_depth_memory(int w, int h){
    hipMalloc((void**)&depth_pixels_cuda, w * h * sizeof(unsigned short));  
}


// color, albedo, normal, fmv, id
void fix_depth(int w, int h, unsigned short * depth_pixels){
    hipMemcpy(depth_pixels_cuda, depth_pixels, sizeof(unsigned short) * w * h, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->albedo, host_pictures[1], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->normal, host_pictures[2], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->fmv, host_pictures[3], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->id, host_pictures[4], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_mask->angular_gaze, gaze_data, sizeof(float) * h_mask->m_gaze_length * 2, hipMemcpyHostToDevice);

    dim3 block(16, 16);  
    dim3 grid((w + block.x - 1) / block.x,   
              (h + block.y - 1) / block.y);  

    for(int iterate_num = 0; iterate_num < 15; iterate_num++){
        convert_4_to_channels<<<grid, block>>>(w, h, depth_pixels_cuda);  
    }
    hipDeviceSynchronize();  
    hipMemcpy(depth_pixels, depth_pixels_cuda, sizeof(unsigned short) * w * h, hipMemcpyDeviceToHost);
}  