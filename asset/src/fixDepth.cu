#include "hip/hip_runtime.h"
#include "fixDepth.h"

__global__ static void convert_4_to_channels(int w, int h) {  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    int idy = blockIdx.y * blockDim.y + threadIdx.y;  
    int pixel_pos = idy * w + idx;
    if (idx < w && idy < h) {  
        return;
    }  
}  


unsigned short * depth_pixels_cuda;

void allocate_fix_depth_memory(int w, int h){
    // hipMalloc((void**)&depth_pixels_cuda, w * h * sizeof(unsigned short));  
}


// color, albedo, normal, fmv, id
void fix_depth(int w, int h, unsigned short * depth_pixels){
    // hipMemcpy(depth_pixels_cuda, depth_pixels, sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->albedo, host_pictures[1], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->normal, host_pictures[2], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->fmv, host_pictures[3], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_cuda_pictures_4->id, host_pictures[4], sizeof(float) * w * h * 4, hipMemcpyHostToDevice);
    // hipMemcpy(h_mask->angular_gaze, gaze_data, sizeof(float) * h_mask->m_gaze_length * 2, hipMemcpyHostToDevice);

    dim3 block(16, 16);  
    dim3 grid((w + block.x - 1) / block.x,   
              (h + block.y - 1) / block.y);  
    
    // convert_4_to_channels<<<grid, block>>>(w, h);  
    // hipDeviceSynchronize();  
}  